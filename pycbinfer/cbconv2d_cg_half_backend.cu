//Copyright (c) 2018 ETH Zurich, Lukas Cavigelli
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

using namespace std;
typedef __half half;


   __global__ void changeDetection_1x1_kernel(
                                         const half* __restrict__ input,
                                         half* __restrict__ inputState,
                                         bool* __restrict__ changeMap,
                                         const int width, const int height, const int nInputPlane,
                                         const float diffThreshold_float,
                                         const bool updateInputState) {

    // compute pixel index
    int pxlInpIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pxlInpIdx >= height * width) return;

    //check for changes at the pixel location; any feature map can trigger a change
	half diffThreshold = __float2half(diffThreshold_float);
    bool change = false;
    for (int i = 0; i < nInputPlane; ++i) {
      int idx = i*(height*width) + pxlInpIdx; // !! this expression might need the long datatype for high res
	  half diff = __hsub(inputState[idx], input[idx]);
      change |= __hgt(diff, diffThreshold) | __hlt(diff, __hneg(diffThreshold));
    }

    if(!change) return; // no need to mark outputs/proceed, if not changed

    // mark pixels in the support of the changed pixel for updating
    changeMap[pxlInpIdx] = true;

    //update prevInput if with copyChanges/feedback
    if(updateInputState) { // implicit (&& change), otherwise already returned
      for (int i = 0; i < nInputPlane; ++i) {
        int idx = i*(height*width) + pxlInpIdx; // !! this expression might need the long datatype for high res                                                           
        inputState[idx] = input[idx];
      }
    }
  }
  
  __global__ void changeDetection_kernel(
                                         const half* __restrict__ input,
                                         half* __restrict__ inputState,
                                         bool* __restrict__ changeMap,
                                         const int width, const int height, const int nInputPlane,
                                         const int kHHalf, const int kWHalf, const float diffThreshold_float,
                                         const bool updateInputState) {

    // compute pixel index
    int pxlInpIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pxlInpIdx >= height * width) return;

    //check for changes at the pixel location; any feature map can trigger a change
	half diffThreshold = __float2half(diffThreshold_float);
    bool change = false;
    for (int i = 0; i < nInputPlane; ++i) {
      int idx = i*(height*width) + pxlInpIdx; // !! this expression might need the long datatype for high res
	  half diff = __hsub(inputState[idx], input[idx]);
      change |= __hgt(diff, diffThreshold) | __hlt(diff, __hneg(diffThreshold));
    }

    if(!change) return; // no need to mark outputs/proceed, if not changed

    // mark pixels in the support of the changed pixel for updating
    int xIn = pxlInpIdx % width;
    int yIn = pxlInpIdx / width;
    for (int k = -kHHalf; k <= kHHalf; ++k) {
      int yOut = yIn + k;
      for (int l = -kWHalf; l <= kWHalf; ++l) {
        int xOut = xIn + l;
        if(yOut>=0 && yOut<height && xOut>=0 && xOut<width) {
          changeMap[yOut*width + xOut] = true;
        }
      }
    }

    //update prevInput if with copyChanges/feedback
    if(updateInputState) { // implicit (&& change), otherwise already returned
      for (int i = 0; i < nInputPlane; ++i) {
        int idx = i*(height*width) + pxlInpIdx; // !! this expression might need the long datatype for high res                                                           
        inputState[idx] = input[idx];
      }
    }
  }

extern "C" {
  void changeDetection(int gridz, int gridy, int gridx, int blockz, int blocky, int blockx,
                       const half* __restrict__ input,
                       half* __restrict__ oldinput,
                       bool* __restrict__ changeMap,
                       const int width, const int height, const int nInputPlane,
                       const int kHHalf, const int kWHalf, const float diffThreshold,
                       const bool updateInputState) {
    dim3 grid(gridx, gridy, gridz);
    dim3 block(blockx, blocky, blockz);
    
    if(kHHalf == 0 and kWHalf == 0) {      
      changeDetection_1x1_kernel<<<grid, block>>>(input, oldinput, changeMap, width, height,
                                                  nInputPlane, diffThreshold, updateInputState);
    } else {
      changeDetection_kernel<<<grid, block>>>(input, oldinput, changeMap, width, height, 
                                              nInputPlane, kHHalf, kWHalf, diffThreshold, updateInputState);
    }
  }
  __global__ void changePropagation_kernel(const bool* __restrict__ changeMatrixIn,
                                           bool* __restrict__ changeMatrixOut,
                                           const int width, const int height,
                                           const int kHHalf, const int kWHalf) {

    // compute pixel index
    int pxlInpIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pxlInpIdx >= height * width) return;

    //check if any pixel in the input range has changed
    bool change = false;
    int xOut = pxlInpIdx % width;
    int yOut = pxlInpIdx / width;
    for (int k = -kHHalf; k <= kHHalf; ++k) {
      int yIn = yOut + k;
      for (int l = -kWHalf; l <= kWHalf; ++l) {
        int xIn = xOut + l;
        if(yIn>=0 && yIn<height && xIn>=0 && xIn<width) {
          change = change || changeMatrixIn[yIn*width + xIn];
        }
      }
    }
    changeMatrixOut[yOut*width + xOut] = change;
  }

  void changePropagation(int gridz, int gridy, int gridx, int blockz, int blocky, int blockx,
                         const bool* __restrict__ changeMatrixIn,
                         bool* __restrict__ changeMatrixOut,
                         const int width, const int height,
                         const int kHHalf, const int kWHalf) {
    dim3 grid(gridx, gridy, gridz);
    dim3 block(blockx, blocky, blockz);
    
    changePropagation_kernel<<<grid, block>>>(changeMatrixIn, changeMatrixOut, 
                                              width, height, kHHalf, kWHalf);
  }

  __global__ void genXMatrix_kernel(
                                    half* columns,
                                    const half* __restrict__ input,
                                    const int* __restrict__ changeList,
                                    const int kW, const int kH,
                                    const int nInputPlane, const int width, const int height,
                                    const int numChanges) {

    const int kx = threadIdx.x;
    const int ky = threadIdx.z;
    const	int changeIdx = blockIdx.x * blockDim.y + threadIdx.y;

    if(changeIdx < numChanges) {
      int pos = changeList[changeIdx];
      int ix = pos % width + kx - (kW-1)/2;
      int iy = pos / width + ky - (kH-1)/2;
      half *dst = columns + changeIdx*(kW*kH*nInputPlane) + ky*kW+kx;

      const bool isInImage = ix>=0 && ix < width && iy>=0 && iy< height;
      for (int i = 0; i < nInputPlane; ++i) {
        dst[i*kH*kW] = isInImage ? input[(i*height + iy) * width + ix] : __float2half(0.0f);
      }
    }
  }

  void genXMatrix(int gridz, int gridy, int gridx, int blockz, int blocky, int blockx,
                  half* columns,
                  const half* __restrict__ input,
                  const int* __restrict__ changeList,
                  const int kW, const int kH,
                  const int nInputPlane, const int width, const int height,
                  const int numChanges) {
    dim3 grid(gridx, gridy, gridz);
    dim3 block(blockx, blocky, blockz);
    genXMatrix_kernel<<<grid, block>>>(columns, input, changeList, kW, kH, nInputPlane, width, height, numChanges);
  }

  __global__ void updateOutput_kernel(const half* __restrict__ columnsOut,
                                      half* output, const int* __restrict__ changeList,
                                      const int numOutputPixel, const int numChanges, const int nOutputPlane, const bool relu) {

    int count = blockIdx.x * blockDim.x + threadIdx.x;

    if(count < numChanges*nOutputPlane) {
      int outpPlane = count / numChanges;
      int changeNr =  count % numChanges;
      int pxl = changeList[changeNr];
      half v = columnsOut[count];
      v = relu && __hle(v, __float2half(0.0f)) ? __float2half(0.0f) : v;
      output[outpPlane*numOutputPixel + pxl] = v;
    }
  }

  void updateOutput(int gridz, int gridy, int gridx, int blockz, int blocky, int blockx,
                    half *columnsOut, half *output, int* changeList, 
                    int numOutputPixel, int numChanges, int nOutputPlane, bool relu) {
    dim3 grid(gridx, gridy, gridz);
    dim3 block(blockx, blocky, blockz);
    updateOutput_kernel<<<grid, block>>>(columnsOut, output, changeList, numOutputPixel, numChanges, nOutputPlane, relu);
  }

  __global__ void maxPool2d_kernel(const half* __restrict__ input,
                                   half* __restrict__ output, 
                                   const int* __restrict__ changeIndexes,
                                   const int numChanges, 
                                   const int numCh, const int iheight, const int iwidth,
                                   const int oheight, const int owidth,
                                   const int stridey, const int stridex) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= numChanges)
      return;
    int pxIdx = changeIndexes[tid];
    int y = pxIdx / iwidth, x = pxIdx % iwidth;
    int yo = y / stridey, xo = x / stridex;

    for(int ch = 0; ch < numCh; ch++) {
      half v = __float2half(-INFINITY);
      for(int j = 0; j < stridey; j++) {
        for(int i = 0; i < stridex; i++) {
          int yi = yo*stridey + j, xi = xo*stridex + i;
          if(yi < iheight && xi < iwidth) {
			half iVal = input[(ch*iheight + yi)*iwidth + xi];
			if (__hgt(iVal, v))
				v = iVal;
          }
        }
      }
      output[(ch*oheight + yo)*owidth + xo] = v;
    }
    
  }

  void maxPool2d(int gridx, int blockx,
                 half *input, half *output, int* changeIndexes, int numChanges, 
                 int numCh, int iheight, int iwidth,
                 int oheight, int owidth,
                 int stridey, int stridex) {
                 
    //determine grid and block size based on number of changes
    dim3 grid(gridx);
    dim3 block(blockx);
    maxPool2d_kernel<<<grid, block>>>(input, output, changeIndexes, numChanges, 
                                      numCh, iheight, iwidth, oheight, owidth, stridey, stridex);
  }

}
